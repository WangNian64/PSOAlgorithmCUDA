#include "hip/hip_runtime.h"
#pragma once
#include <iostream>
#include <fstream>
#include <string>
#include <iomanip>
#include <ctime>

#include "FitnessFunction.h"
int main()
{
	int problemSum = 1;//�������Ŀ
	int testSum = 1;//ÿ��ʵ���ܵ�ʵ�����
	for (int curProblem = 0; curProblem < problemSum; curProblem++)//�ܶ������
	{
		cout << "�ܵ�" + to_string(curProblem + 1) + "������" << endl;

#pragma region ����PSO����
		ifstream inputFile;
		inputFile.open("../../InputParas/InputPara" + to_string(curProblem + 1) + ".txt");

		//CPU
		ProblemParas proParas(inputFile);					// ��ʼ�������豸��ز��� 
		cout << "�ļ���ȡ�ɹ�" << endl;
		int ThreadsPerBlock = 100;							// һ��Block��100��thread
		int BlockSum = 1;									// Block����Ŀ
		int dim = proParas.DeviceSum * 3;					// ��ά��=�豸��*3(x,y,����)
		//CPU
		PSOPara psopara(dim);
		psopara.mesh_div_count = 4;							// ���񻮷���Ŀ
		psopara.problemParas = proParas;					// ��������Ĳ���
		psopara.particle_num_ = ThreadsPerBlock * BlockSum;	// ���Ӹ���
		psopara.max_iter_num_ = 400;						// ����������
		psopara.fitness_count_ = 2;							// ��Ӧ����Ŀ
		psopara.archive_max_count = 50;						// archive����������Ŀ
		psopara.SetDt(1.0);									// ʱ�䲽��
		psopara.SetWstart(0.9);								// ��ʼȨ��
		psopara.SetWend(0.4);								// ����Ȩ��
		psopara.SetC1(1.49445);								// ���ٶ�����1
		psopara.SetC2(1.49445);								// ���ٶ�����2
		psopara.SetLowBound(0, 0, DeviceDirect::Default);	// position��������Χ����

		psopara.blockSum = BlockSum;
		psopara.threadsPerBlock = ThreadsPerBlock;
		//��Ҫ���豸����ȡ�����ֵ��ֻ��ȡ��3.��
		psopara.SetUpBound(proParas.workShopLength, proParas.workShopWidth, DeviceDirect::Rotate270 + 1);// position��������Χ����
#pragma endregion

#pragma region ����PSO�㷨����������
//GPU
		PSOOptimizer psooptimizer(&psopara, proParas);//PSO�㷨����
		string curProblemFolderName = "Problem" + to_string(curProblem + 1);
		for (int curTest = 0; curTest < testSum; curTest++) {//ÿ�������ܶ��
			clock_t startTime, endTime;//��¼����ʱ��

			startTime = clock();//��ʱ��ʼ
#pragma region ��ʼ��
			psooptimizer.InitialAllParticles();//��ʼ���������� CPU
			psooptimizer.InitialArchiveList();//��ʼ��Archive�浵	CPU
			psooptimizer.InitGbest();//��ʼ��ȫ������		CPU
#pragma endregion

#pragma region ������������&��ÿһ�ε���Ӧ��ֵ
//Ŀ��1��ֵ����archiveList1�У�Ŀ��2��ֵ����archiveList2��
//��n��ʵ��ŵ��ļ���ȥ
//�ļ��е����ֽ�Testn
			ofstream OutFile;
			ofstream OutFile1;
			string curTestFolderName = "Test" + to_string(curTest + 1);
			OutFile.open("../../Results/" + curProblemFolderName + "/" + curTestFolderName + "/archiveList1.txt");
			OutFile1.open("../../Results/" + curProblemFolderName + "/" + curTestFolderName + "/archiveList2.txt");
			for (int i = 0; i < psooptimizer.max_iter_num_; i++)//��ʼ���в���
			{
				cout << (i + 1) << endl;
				psooptimizer.UpdateAllParticles();//�����������ӵ�λ�ú��ٶ�
				psooptimizer.UpdatePbest();//����pbest 

				psooptimizer.UpdateArchiveList();//�����ⲿ�浵����
				psooptimizer.UpdateGbest();//����gbest

				//�洢ÿ�ε�����Archive����
				double minFitness1, minFitness2;
				minFitness1 = minFitness2 = INT_MAX;
				cout << minFitness1 << endl;
				//archiveList��CPU��
				for (auto it = psooptimizer.archive_list.begin(); it != psooptimizer.archive_list.end(); it++)
				{
					minFitness1 = min(minFitness1, it->fitness_[0]);
				}
				string f1line = to_string(minFitness1) + "\n";
				OutFile << f1line;
				cout << f1line << endl;

				for (auto it = psooptimizer.archive_list.begin(); it != psooptimizer.archive_list.end(); it++)
				{
					minFitness2 = min(minFitness2, it->fitness_[1]);
				}
				string f2line = to_string(minFitness2) + "\n";
				OutFile1 << f2line;
				cout << f2line << endl;
			}
			OutFile.close();
			OutFile1.close();
#pragma endregion

			endTime = clock();
			cout << "����" << psopara.max_iter_num_ << "�ε�������ʱ:" << static_cast<double>(endTime - startTime) / CLOCKS_PER_SEC << "s" << endl;

#pragma region �����豸�ߴ�&���ղ��ֽ��&���ߵ������
			OutFile.open("../../Results/" + curProblemFolderName + "/" + curTestFolderName + "/FinalResult.txt");




#pragma region ��¼���ղ��ֽ��
			int resultIndex = 0;
			int minHandleCost = INT_MAX;
			int minConveyValue = INT_MAX;
			//����ѡ��������ɱ���͵�
			for (int i = 0; i < psooptimizer.archive_list.size(); i++)
			{
				if (psooptimizer.archive_list[i].fitness_[0] < minHandleCost)
				{
					minHandleCost = psooptimizer.archive_list[i].fitness_[0];
					resultIndex = i;
				}
			}
			//����ѡ���ͻ��ɱ��͵�
			//for (int i = 0; i < psooptimizer.archive_list.size(); i++)
			//{
			//	if (psooptimizer.archive_list[i].fitness_[1] < minConveyValue)
			//	{
			//		minConveyValue = psooptimizer.archive_list[i].fitness_[0];
			//		resultIndex = i;
			//	}
			//}

			for (int i = 0; i < dim; i += 3)
			{
				OutFile << psooptimizer.archive_list[resultIndex].position_[i];
				OutFile << ",";
				OutFile << psooptimizer.archive_list[resultIndex].position_[i + 1];
				OutFile << "\n";
			}
#pragma endregion

			//������һЩ������Ҫ��GPU->CPU
			//ֻ������Ҫ��
#pragma region ��¼�豸�ߴ�
			Vector2* deviceParaListSize_CPU = new Vector2[proParas.DeviceSum];
			//��ѭ���ķ�ʽ���������ݵ�CPU��ÿ��ֻ����һ��size�ĵ�ַ
			for (int i = 0; i < proParas.DeviceSum; i++)
			{
				hipMemcpy(deviceParaListSize_CPU + i, &psooptimizer.problemParas.size[i], sizeof(Vector2), hipMemcpyDeviceToHost);
			}

			for (int i = 2; i < dim; i += 3)
			{
				DeviceDirect direct = (DeviceDirect)(int)psooptimizer.archive_list[resultIndex].position_[i];
				string line = "";
				if (direct == DeviceDirect::Rotate90 || direct == DeviceDirect::Rotate270)
				{
					//�����������GPU�ϵ�
					line = to_string(deviceParaListSize_CPU[i / 3].y) + "," +
						to_string(deviceParaListSize_CPU[i / 3].x);
				}
				else {
					line = to_string(deviceParaListSize_CPU[i / 3].x) + "," +
						to_string(deviceParaListSize_CPU[i / 3].y);
				}
				OutFile << line + "\n";
			}
#pragma endregion

			int fitnessIndex = 0;
#pragma region ��¼��������꣨��ת֮��ģ������豸���꣩

			//����һ��inoutPoints
			//int ioPointsSize = psooptimizer.bestPathInfoList[fitnessIndex].inoutPSize;
			int ioPointsSize = psooptimizer.inoutPSize;//
			InoutPoint* ioPoints = new InoutPoint[ioPointsSize];
			hipMemcpy(ioPoints, psooptimizer.curBestPath_InoutPoints, sizeof(InoutPoint) * ioPointsSize, hipMemcpyDeviceToHost);

			OutFile << to_string(ioPointsSize) + "\n";//�������Ŀ
			for (int i = 0; i < ioPointsSize; i++)
			{
				if (ioPoints[i].pointDirect == PointDirect::Up || ioPoints[i].pointDirect == PointDirect::Down)
				{
					OutFile << "Vertical ";
				}
				else
				{
					OutFile << "Horizon ";
				}
				OutFile << ioPoints[i].pointAxis.x;
				OutFile << " ";
				OutFile << ioPoints[i].pointAxis.y;
				OutFile << "\n";
			}
#pragma endregion

#pragma region ��¼�����·��
			////�ȴ�ÿ�ֻ����·������
			//string line = "";
			//for (int i = 0; i < proParas.CargoTypeNum; i++)
			//{
			//	line += to_string(proParas.cargoTypeList[i].deviceSum - 1);
			//	if (i != proParas.CargoTypeNum - 1)
			//	{
			//		line += " ";
			//	}
			//}
			//OutFile << line << "\n";

			//vector<PointLink> p = psooptimizer.archive_list[resultIndex].pointLinks;
			//for (int i = 0; i < p.size(); i++)
			//{
			//	string s1, s2;
			//	DevicePara device1, device2;

			//	//s1 = to_string(p[i].device1Index) + " " + to_string(p[i].device2Index);
			//	OutFile << to_string(p[i].device1Index) + " " + to_string(p[i].device2Index) + " ";
			//	//����s2
			//	for (int j = 0; j < p[i].points.size(); j++)
			//	{
			//		OutFile /*<< fixed << setprecision(1)*/ << p[i].points[j].x;
			//		OutFile << ",";
			//		OutFile /*<< fixed << setprecision(1)*/ << p[i].points[j].y;
			//		//s2 += to_string(p[i].points[j].x) + "," + to_string(p[i].points[j].y);
			//		if (j != p[i].points.size() - 1)
			//		{
			//			//s2 += "|";
			//			OutFile << "|";
			//		}
			//	}
			//	OutFile << "\n";
			//	//string line = s1 + " " + s2 + "\n";
			//	//OutFile << line;
			//}
#pragma endregion

#pragma region ��¼ֱ�����ͻ���ת�����ͻ�����
//GPU->CPU
			int strInfoListSum = psooptimizer.curBestPath_StrConveyorListSum[0];
			StraightConveyorInfo* strInfoList = new StraightConveyorInfo[strInfoListSum];
			hipMemcpy(strInfoList, psooptimizer.curBestPath_StrConveyorList, sizeof(StraightConveyorInfo) * strInfoListSum, hipMemcpyDeviceToHost);

			int curveInfoListSum = psooptimizer.curBestPath_CurveConveyorListSum[0];
			Vector2Int* curveInfoList = new Vector2Int[curveInfoListSum];
			hipMemcpy(curveInfoList, psooptimizer.curBestPath_CurveConveyorList, sizeof(Vector2Int)* curveInfoListSum, hipMemcpyDeviceToHost);

			OutFile << strInfoListSum << "\n";
			for (int i = 0; i < strInfoListSum; i++)
			{
				OutFile << to_string(strInfoList[i].startPos.x) << "," << to_string(strInfoList[i].startPos.y)
					<< ";" << to_string(strInfoList[i].endPos.x) << "," << to_string(strInfoList[i].endPos.y)
					<< ";" << to_string(strInfoList[i].startHnum) << ";" << to_string(strInfoList[i].startVnum)
					<< ";" << to_string(strInfoList[i].endHnum) << ";" << to_string(strInfoList[i].endVnum)
					<< "\n";
			}
			OutFile << curveInfoListSum << "\n";
			for (int i = 0; i < curveInfoListSum; i++)
			{
				OutFile << to_string(curveInfoList[i].x) << "," << to_string(curveInfoList[i].y) << "\n";
			}
#pragma endregion

			OutFile.close();
#pragma endregion

		}

		//�����ڴ�

#pragma endregion
	}
	return 0;
}
