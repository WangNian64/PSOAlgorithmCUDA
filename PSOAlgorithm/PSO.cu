#include "hip/hip_runtime.h"
#pragma once
#include "Pareto.h"
#include "Archive.h"
#include "FitnessFunction.h"
#include "Tools.h"
#include <ctime>


//��дstatic

static __global__ void InitPointDirectArray(int* pointDirectArray)
{
	pointDirectArray[0] = -1;
	pointDirectArray[1] = -1;
	pointDirectArray[2] = -1;
	pointDirectArray[3] = -1;
	pointDirectArray[4] = -1;

	pointDirectArray[5] = -1;
	pointDirectArray[6] = 1;
	pointDirectArray[7] = 2;
	pointDirectArray[8] = 3;
	pointDirectArray[9] = 4;

	pointDirectArray[10] = -1;
	pointDirectArray[11] = 5;
	pointDirectArray[12] = 6;
	pointDirectArray[13] = 7;
	pointDirectArray[14] = 8;

	pointDirectArray[15] = -1;
	pointDirectArray[16] = 9;
	pointDirectArray[17] = 10;
	pointDirectArray[18] = 11;
	pointDirectArray[19] = 12;

	pointDirectArray[20] = -1;
	pointDirectArray[21] = 13;
	pointDirectArray[22] = 14;
	pointDirectArray[23] = 15;
	pointDirectArray[24] = 16;
}
static __global__ void InitCurBestPathFit(double* curBestPath_FitnessVal)
{
	curBestPath_FitnessVal[0] = 1000000;
}

//��������&������Ӧ��
static __global__ void UpdateParticle_Kernal(int curIterNum, int maxIterNum, int dim, int fitnessCount, double w_, double C1_, double C2_, double dt_,
	/*�������Particle Particle* particles_,*/double* fitness_GPU, double* position_GPU, double* velocity_GPU, double* best_position_GPU, double* best_fitness_GPU,
	hiprandState* globalState, double* randomNumList, double* range_interval_, double* upper_bound_, double* lower_bound_, double* all_best_position_,
	/*ProblemParas problemParas*/Vector2* size, double* spaceLength, double workShopLength, double workShopWidth)
{
	//���ӵ��±�i��Ҫ�Լ�����
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	//�ȸ��³���Ȼ����ݳ���������ӵķ�Χ
	for (int j = 2; j < dim; j += 3)
	{
		double last_position = position_GPU[i * dim + j];

		velocity_GPU[i * dim + j] = w_ * velocity_GPU[i * dim + j] +
			C1_ * createARandomNum(globalState, i) * (best_position_GPU[i * dim + j] - position_GPU[i * dim + j]) +
			C2_ * createARandomNum(globalState, i) * (all_best_position_[i * dim + j] - position_GPU[i * dim + j]);
		position_GPU[i * dim + j] += dt_ * velocity_GPU[i * dim + j];

		// �����������������������
		if (upper_bound_ && lower_bound_)
		{
			if (position_GPU[i * dim + j] >= upper_bound_[j])//ע������豸����=Ҳ����
			{
				double thre = createARandomNum(globalState, i);//ֱ������һ�������
				if (last_position >= upper_bound_[j] - 1)//ע��upper_bound_[j]-1=3
				{
					position_GPU[i * dim + j] = createARandomNum(globalState, i) * range_interval_[j] + lower_bound_[j];
				}
				else if (thre < 0.5)
				{
					position_GPU[i * dim + j] = upper_bound_[j] - (upper_bound_[j] - last_position) * createARandomNum(globalState, i);
				}
				else
				{
					position_GPU[i * dim + j] = upper_bound_[j] - 0.5;
				}
			}
			if (position_GPU[i * dim + j] < lower_bound_[j])
			{
				double thre = createARandomNum(globalState, i);
				if (last_position == lower_bound_[j])
				{
					position_GPU[i * dim + j] = createARandomNum(globalState, i) * range_interval_[j] + lower_bound_[j];
				}
				else if (thre < 0.5)
				{
					position_GPU[i * dim + j] = lower_bound_[j] + (last_position - lower_bound_[j]) * createARandomNum(globalState, i);
				}
				else
				{
					position_GPU[i * dim + j] = lower_bound_[j];
				}
			}
		}
	}
	//���ݳ����޸��豸���½緶Χ
	for (int j = 2; j < dim; j += 3)
	{
		//doubleתint��ת��ΪDirection��Ȼ����ݳ������¼����豸�ߴ�ͳ����
		//Rotate90����Rotate270,�޸�������
		DeviceDirect curDirect = (DeviceDirect)(int)position_GPU[i * dim + j];
		if (curDirect == DeviceDirect::Rotate90 || curDirect == DeviceDirect::Rotate270)//��һ���ֿ���ҲҪ�ģ�enum��C++�﷨��
		{
			//x��y
			lower_bound_[j - 2] = 0 + size[j / 3].y * 0.5 + spaceLength[j / 3];
			lower_bound_[j - 1] = 0 + size[j / 3].x * 0.5 + spaceLength[j / 3];

			upper_bound_[j - 2] = workShopLength - size[j / 3].y * 0.5 - spaceLength[j / 3];
			upper_bound_[j - 1] = workShopWidth - size[j / 3].x * 0.5 - spaceLength[j / 3];

		}
		else
		{
			//x��y
			lower_bound_[j - 2] = 0 + size[j / 3].x * 0.5 + spaceLength[j / 3];
			lower_bound_[j - 1] = 0 + size[j / 3].y * 0.5 + spaceLength[j / 3];

			upper_bound_[j - 2] = workShopLength - size[j / 3].x * 0.5 - spaceLength[j / 3];
			upper_bound_[j - 1] = workShopWidth - size[j / 3].y * 0.5 - spaceLength[j / 3];

		}
		range_interval_[j - 2] = upper_bound_[j - 2] - lower_bound_[j - 2];
		range_interval_[j - 1] = upper_bound_[j - 1] - lower_bound_[j - 1];
	}
	//cout << endl;
	for (int j = 0; j < dim; j++)
	{
		if (j % 3 != 2)
		{
			//������һ�ε��������position��velocity
			double last_position = position_GPU[i * dim + j];

			velocity_GPU[i * dim + j] = w_ * velocity_GPU[i * dim + j] +
				C1_ * createARandomNum(globalState, i) * (best_position_GPU[i * dim + j] - position_GPU[i * dim + j]) +
				C2_ * createARandomNum(globalState, i) * (all_best_position_[i * dim + j] - position_GPU[i * dim + j]);
			position_GPU[i * dim + j] += dt_ * velocity_GPU[i * dim + j];

			// �����������������������
			if (upper_bound_ && lower_bound_)
			{
				if (position_GPU[i * dim + j] > upper_bound_[j])
				{
					double thre = createARandomNum(globalState, i);
					if (last_position >= upper_bound_[j])
					{
						position_GPU[i * dim + j] = createARandomNum(globalState, i) * range_interval_[j] + lower_bound_[j];
					}
					else if (thre < 0.5)
					{
						position_GPU[i * dim + j] = upper_bound_[j] - abs(upper_bound_[j] - last_position) * createARandomNum(globalState, i);
					}
					else
					{
						position_GPU[i * dim + j] = upper_bound_[j];
					}
				}
				if (position_GPU[i * dim + j] < lower_bound_[j])
				{
					double thre = createARandomNum(globalState, i);
					if (last_position <= lower_bound_[j])
					{
						position_GPU[i * dim + j] = createARandomNum(globalState, i) * range_interval_[j] + lower_bound_[j];
					}
					else if (thre < 0.5)
					{
						position_GPU[i * dim + j] = lower_bound_[j] + abs(last_position - lower_bound_[j]) * createARandomNum(globalState, i);
					}
					else
					{
						position_GPU[i * dim + j] = lower_bound_[j];
					}
				}
			}
		}

	}
}
// �Ƚ��������ӵ���Ӧ�ȣ��ж��Ƿ���ȫ֧�䣬�Ӷ������pbest
static __device__ bool ComparePbest(int index, int fitness_count, double* fitness, double* pbestFitness, hiprandState* globalState)
{
	int numGreater = 0;
	int numLess = 0;
	for (int i = 0; i < fitness_count; i++)
	{
		if (fitness[i] < pbestFitness[i])
		{
			numGreater++;
		}
		if (fitness[i] > pbestFitness[i])
		{
			numLess++;
		}
	}
	//�����ǰ֧����ʷ������
	if (numGreater > 0 && numLess == 0)
	{
		return true;
	}
	//�����ʷ֧�䵱ǰ���ӣ�������
	else if (numGreater == 0 && numLess > 0)
	{
		return false;
	}
	//�������֧�䣬���ѡ����Ӧ��1�ĸ��ʸߵ㣩
	else
	{
		double randomProb = createARandomNum(globalState, index);//�������С��
		if (fitness[0] < pbestFitness[0])
		{
			if (randomProb < 0.75)
			{
				return true;
			}
			else
			{
				return false;
			}
		}
		else
		{
			if (randomProb > 0.5)
			{
				return true;
			}
			else
			{
				return false;
			}
		}
	}
}

//����Pbest��GPU����
static __global__ void UpdatePbest_Kernal(int dim_, int fitness_count, double* fitness_GPU, double* position_GPU,
	double* velocity_GPU, double* best_position_GPU, double* best_fitness_GPU, hiprandState* globalState)
{
	//i��Ҫ����
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	//�Ƚ���ʷpbest�͵�ǰ��Ӧ�ȣ������Ƿ�Ҫ����
	if (ComparePbest(i, fitness_count, fitness_GPU + i * fitness_count, best_fitness_GPU + i * fitness_count, globalState));
	{
		for (int j = 0; j < fitness_count; j++)
		{
			best_fitness_GPU[i * fitness_count + j] = fitness_GPU[i * fitness_count + j];
			//particles_[i].best_fitness_[j] = particles_[i].fitness_[j];
		}
		for (int j = 0; j < dim_; j++)
		{
			best_position_GPU[i * dim_ + j] = position_GPU[i * dim_ + j];
			//particles_[i].best_position_[j] = particles_[i].position_[j];
		}
	}
}

//����Gbest��GPU����
static __global__ void UpdateGbest_Kernal(int fitness_count, int dim_, double* all_best_fitness_, double* all_best_position_, Particle* gbestList)
{
	//�±��Լ���
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	//����
	for (int j = 0; j < fitness_count; j++)
	{
		all_best_fitness_[i * fitness_count + j] = gbestList[i].best_fitness_[j];
	}
	for (int k = 0; k < dim_; k++)
	{
		all_best_position_[i * dim_ + k] = gbestList[i].best_position_[k];
	}
}





// ���캯��(��ʼ�������㷨�Ĳ��������������ռ�)
PSOOptimizer::PSOOptimizer(PSOPara* pso_para, ProblemParas& problemParas)
{
	blockNum = pso_para->blockSum;
	threadsPerBlock = pso_para->threadsPerBlock;

	particle_num_ = pso_para->particle_num_;
	max_iter_num_ = pso_para->max_iter_num_;
	dim_ = pso_para->dim_;
	fitness_count = pso_para->fitness_count_;
	curr_iter_ = 0;
	meshDivCount = pso_para->mesh_div_count;
	archiveMaxCount = pso_para->archive_max_count;

	//���Ӳ���CPU 
	position_CPU = (double*)malloc(sizeof(double) * particle_num_ * dim_);
	velocity_CPU = (double*)malloc(sizeof(double) * particle_num_ * dim_);
	best_position_CPU = (double*)malloc(sizeof(double) * particle_num_ * dim_);
	fitness_CPU = (double*)malloc(sizeof(double) * particle_num_ * fitness_count);
	best_fitness_CPU = (double*)malloc(sizeof(double) * particle_num_ * fitness_count);

	lower_bound_CPU = (double*)malloc(sizeof(double) * dim_);
	upper_bound_CPU = (double*)malloc(sizeof(double) * dim_);
	range_interval_CPU = (double*)malloc(sizeof(double) * dim_);
	//��ֵ
	for (int i = 0; i < dim_; i++)
	{
		lower_bound_CPU[i] = pso_para->lower_bound_[i];
		upper_bound_CPU[i] = pso_para->upper_bound_[i];
		range_interval_CPU[i] = upper_bound_CPU[i] - lower_bound_CPU[i];
	}


	//���Ӳ���GPU
	hipMalloc((void**)& position_GPU, sizeof(double) * particle_num_ * dim_);
	hipMalloc((void**)& velocity_GPU, sizeof(double) * particle_num_ * dim_);
	hipMalloc((void**)& best_position_GPU, sizeof(double) * particle_num_ * dim_);
	hipMalloc((void**)& fitness_GPU, sizeof(double) * particle_num_ * fitness_count);
	hipMalloc((void**)& best_fitness_GPU, sizeof(double) * particle_num_ * fitness_count);


	hipMalloc((void**)& upper_bound_, sizeof(double) * dim_);
	hipMalloc((void**)& lower_bound_, sizeof(double) * dim_);
	hipMalloc((void**)& range_interval_, sizeof(double) * dim_);
	//����CPU->GPU
	hipMemcpy(upper_bound_, pso_para->upper_bound_, sizeof(double) * dim_, hipMemcpyHostToDevice);
	hipMemcpy(lower_bound_, pso_para->lower_bound_, sizeof(double) * dim_, hipMemcpyHostToDevice);
	hipMemcpy(range_interval_, pso_para->range_interval_, sizeof(double) * dim_, hipMemcpyHostToDevice);


	hipMalloc((void**)& all_best_position_, sizeof(double) * particle_num_ * dim_);
	hipMalloc((void**)& all_best_fitness_, sizeof(double) * particle_num_ * fitness_count);

	//��ʼ�����������
	hipMalloc(&globalState, particle_num_ * sizeof(hiprandState));
	initRandomGenerator << < 1, particle_num_ >> > (globalState, unsigned(time(NULL)));
	//��������������GPU�ռ�
	hipMalloc((void**)& randomNumList, sizeof(double) * particle_num_);

	hipMalloc((void**)& bestParticleIndex, sizeof(int));

	dt_ = pso_para->dt_;
	wstart_ = pso_para->wstart_;
	wend_ = pso_para->wend_;
	C1_ = pso_para->C1_;
	C2_ = pso_para->C2_;

	this->problemParas = problemParas;//CPU




	//��problemParas�Ĳ�����ֵ��GPU��
	DeviceSum = problemParas.DeviceSum;									//�豸����
	horiPointCount = problemParas.horiPointCount;						//δȥ��ǰ����ˮƽ����ĵ����Ŀ
	vertPointCount = problemParas.vertPointCount;						//δȥ��ǰ���д�ֱ����ĵ����Ŀ
	workShopLength = problemParas.workShopLength;						//���䳤��
	workShopWidth = problemParas.workShopWidth;							//������
	entrancePos = problemParas.entrancePos;								//�ֿ��������	
	exitPos = problemParas.exitPos;										//�ֿ��������

	//���ϲ����б�
	fixedLinkPointSum = 50;
	fixedUniqueLinkPointSum = 20;
	//���ͻ�����
	convey2DeviceDist = problemParas.convey2DeviceDist;//���ͻ����豸�ľ��루Ѱ·��ʱ��Ҫ���ǣ�
	conveyWidth = problemParas.conveyWidth;//���ͻ����
	conveyMinLength = problemParas.conveyMinLength;//���ͻ���̳���
	conveySpeed = problemParas.conveySpeed;//���ͻ������ٶ�
	strConveyorUnitCost = problemParas.strConveyorUnitCost;//��λֱ�����ͻ��ɱ�
	curveConveyorUnitCost = problemParas.curveConveyorUnitCost;//����ת�����ͻ��ɱ�
	conveyMinDist = problemParas.conveyMinDist;//������������֮�����̾���



	//DevicePara* deviceParaList;										//�豸�����б�
	//��Ŀ��DeviceSum
	hipMalloc((void**)& ID, sizeof(int)* DeviceSum);
	hipMalloc((void**)& workSpeed, sizeof(double)* DeviceSum);
	hipMalloc((void**)& size, sizeof(Vector2)* DeviceSum);
	hipMalloc((void**)& axis, sizeof(Vector2)* DeviceSum);
	hipMalloc((void**)& direct, sizeof(DeviceDirect)* DeviceSum);
	hipMalloc((void**)& spaceLength, sizeof(double)* DeviceSum);
	//����ڵ�����飨��Ӱ�������ߵĲ��֣�
	hipMalloc((void**)& adjPInCount, sizeof(int)* DeviceSum);
	hipMalloc((void**)& adjPOutCount, sizeof(int)* DeviceSum);
	hipMalloc((void**)& accumAdjPInCount, sizeof(int)* DeviceSum);
	hipMalloc((void**)& accumAdjPOutCount, sizeof(int)* DeviceSum);

	totalInPoint = problemParas.totalInPoint;
	totalOutPoint = problemParas.totalOutPoint;
	//cout << "totalInPoint:" << totalInPoint << endl;
	//cout << "totalOutPoint:" << totalOutPoint << endl;
	hipMalloc((void**)& adjPointsIn, sizeof(AdjPoint)* totalInPoint);
	hipMalloc((void**)& adjPointsOut, sizeof(AdjPoint)* totalOutPoint);



	//CPU��ֵ��GPU
	hipMemcpy(ID, problemParas.ID, sizeof(int) * DeviceSum, hipMemcpyHostToDevice);
	hipMemcpy(workSpeed, problemParas.workSpeed, sizeof(double) * DeviceSum, hipMemcpyHostToDevice);
	hipMemcpy(size, problemParas.size, sizeof(Vector2) * DeviceSum, hipMemcpyHostToDevice);
	hipMemcpy(axis, problemParas.axis, sizeof(Vector2) * DeviceSum, hipMemcpyHostToDevice);
	hipMemcpy(direct, problemParas.direct, sizeof(DeviceDirect) * DeviceSum, hipMemcpyHostToDevice);
	hipMemcpy(spaceLength, problemParas.spaceLength, sizeof(double) * DeviceSum, hipMemcpyHostToDevice);

	hipMemcpy(adjPInCount, problemParas.adjPInCount, sizeof(int) * DeviceSum, hipMemcpyHostToDevice);
	hipMemcpy(adjPOutCount, problemParas.adjPOutCount, sizeof(int) * DeviceSum, hipMemcpyHostToDevice);
	hipMemcpy(accumAdjPInCount, problemParas.accumAdjPInCount, sizeof(int) * DeviceSum, hipMemcpyHostToDevice);
	hipMemcpy(accumAdjPOutCount, problemParas.accumAdjPOutCount, sizeof(int) * DeviceSum, hipMemcpyHostToDevice);

	hipMemcpy(adjPointsIn, problemParas.adjPointsIn, sizeof(int) * totalInPoint, hipMemcpyHostToDevice);
	hipMemcpy(adjPointsOut, problemParas.adjPointsOut, sizeof(int) * totalOutPoint, hipMemcpyHostToDevice);


	//CargoType* cargoTypeList;						//���������б�
	//��Ŀ��CargoTypeNum
	CargoTypeNum = problemParas.CargoTypeNum;		//����������Ŀ
	totalLinkSum = problemParas.totalLinkSum;		//�ܵ���������Ŀ

	hipMalloc((void**)& deviceSum, sizeof(int)* CargoTypeNum);
	hipMalloc((void**)& linkSum, sizeof(int)* CargoTypeNum);
	hipMalloc((void**)& accumLinkSum, sizeof(int)* CargoTypeNum);
	hipMalloc((void**)& totalVolume, sizeof(double)* CargoTypeNum);
	hipMalloc((void**)& deviceLinkList, sizeof(DeviceLink)* totalLinkSum);
	//CPU��ֵ��GPU
	hipMemcpy(deviceSum, problemParas.deviceSum, sizeof(int)* CargoTypeNum, hipMemcpyHostToDevice);
	hipMemcpy(linkSum, problemParas.linkSum, sizeof(int)* CargoTypeNum, hipMemcpyHostToDevice);
	hipMemcpy(accumLinkSum, problemParas.accumLinkSum, sizeof(int)* CargoTypeNum, hipMemcpyHostToDevice);
	hipMemcpy(deviceLinkList, problemParas.deviceLinkList, sizeof(DeviceLink)* totalLinkSum, hipMemcpyHostToDevice);
	hipMemcpy(totalVolume, problemParas.totalVolume, sizeof(double)* CargoTypeNum, hipMemcpyHostToDevice);



	/*�������ӵ���������Ϣ�洢*/
	//��Ŀ��particle_num_
	//GPU�ڴ����
	hipMalloc((void**)& curBestFitnessVal, sizeof(double) * particle_num_);
	inoutPSize = totalInPoint + totalOutPoint;
	//hipMalloc((void**)& inoutPSize, sizeof(int) * particle_num_);
	hipMalloc((void**)& inoutPoints, sizeof(InoutPoint) * inoutPSize * particle_num_);
	hipMalloc((void**)& strConveyorList, sizeof(StraightConveyorInfo) * fixedUniqueLinkPointSum * totalLinkSum * particle_num_);
	hipMalloc((void**)& strConveyorListSum, sizeof(int) * particle_num_);
	hipMalloc((void**)& curveConveyorList, sizeof(Vector2Int) * fixedUniqueLinkPointSum * totalLinkSum * particle_num_);
	hipMalloc((void**)& curveConveyorListSum, sizeof(int) * particle_num_);


	//��ǰ�������������Ϣ
	//��Ŀ1(�ȿ�����Ӧ��1��
	//GPU�ڴ����	
	hipMalloc((void**)& curBestPath_FitnessVal, sizeof(double));
	hipMalloc((void**)& curBestPath_InoutPoints, sizeof(InoutPoint) * inoutPSize);
	hipMalloc((void**)& curBestPath_StrConveyorList, sizeof(StraightConveyorInfo)* fixedUniqueLinkPointSum * totalLinkSum);
	hipMalloc((void**)& curBestPath_StrConveyorListSum, sizeof(int));
	hipMalloc((void**)& curBestPath_CurveConveyorList, sizeof(Vector2Int) * fixedUniqueLinkPointSum * totalLinkSum);
	hipMalloc((void**)& curBestPath_CurveConveyorListSum, sizeof(int));
	//��ʼ��
	InitCurBestPathFit << <1, 1 >> > (curBestPath_FitnessVal);


	hipMalloc((void**)& pointDirectArray, sizeof(int) * 25);
	InitPointDirectArray << <1, 1 >> > (pointDirectArray);
}
//���µ�ǰ���������·��Ϣ
//ֻ��Ҫִ��һ��
static __global__ void UpdateCurBestPathInfo(int fitnessCount, double* fitness_GPU, int* bestParticleIndex,
	int tempStrConveyorList_PointSum, int tempCurveConveyorList_PointSum,
	/*���е���������Ϣ*/
	double* curBestFitnessVal, int inoutPSize, InoutPoint* inoutPoints, StraightConveyorInfo* strConveyorList,
	int* strConveyorListSum, Vector2Int* curveConveyorList, int* curveConveyorListSum,
	/*��ѵ���������Ϣ*/
	double* curBestPath_FitnessVal, int curBestPath_InoutPSize, InoutPoint* curBestPath_InoutPoints, StraightConveyorInfo* curBestPath_StrConveyorList,
	int* curBestPath_StrConveyorListSum, Vector2Int* curBestPath_CurveConveyorList, int* curBestPath_CurveConveyorListSum)
{
	//��curbest�����е���������ѱȽϣ�����curBest
	//���Ӧ��ֻҪ����һ��
	//ת������ֻ���ø��Ƶķ�����
	int bestIndex = bestParticleIndex[0];//������Ӷ�Ӧ���±�
	if (curBestPath_FitnessVal[0] > fitness_GPU[bestIndex * fitnessCount + 0]) {
		curBestPath_FitnessVal[0] = fitness_GPU[bestIndex * fitnessCount + 0];//ע��ƫ��ֵ
		for (int i = 0; i < curBestPath_InoutPSize; i++)
		{
			curBestPath_InoutPoints[i] = inoutPoints[bestIndex * inoutPSize + i];
		}
		for (int i = 0; i < strConveyorListSum[bestIndex]; i++) {
			curBestPath_StrConveyorList[i] = strConveyorList[bestIndex * tempStrConveyorList_PointSum + i];
		}
		curBestPath_StrConveyorListSum[0] = strConveyorListSum[bestIndex];
		for (int i = 0; i < curveConveyorListSum[bestIndex]; i++) {
			curBestPath_CurveConveyorList[i] = curveConveyorList[bestIndex * tempCurveConveyorList_PointSum + i];
		}
		curBestPath_CurveConveyorListSum[0] = curveConveyorListSum[bestIndex];
	}
}

PSOOptimizer::~PSOOptimizer()//������������Ҫ�޸�
{
	//CPU
	if (fitness_CPU) { delete[] fitness_CPU; }
	if (position_CPU) { delete[] position_CPU; }
	if (velocity_CPU) { delete[] velocity_CPU; }
	if (best_position_CPU) { delete[] best_position_CPU; }
	if (best_fitness_CPU) { delete[] best_fitness_CPU; }

	if (lower_bound_CPU) { delete[] lower_bound_CPU; }
	if (upper_bound_CPU) { delete[] upper_bound_CPU; }
	if (all_best_position_) { delete[] all_best_position_; }
	//GPU
	hipFree(fitness_GPU);
	hipFree(position_GPU);
	hipFree(velocity_GPU);
	hipFree(best_position_GPU);
	hipFree(best_fitness_GPU);
	hipFree(randomNumList);
	hipFree(globalState);
	hipFree(upper_bound_);
	hipFree(lower_bound_);
	hipFree(range_interval_);
	hipFree(all_best_fitness_);
	hipFree(all_best_position_);
	hipFree(bestParticleIndex);

	//DevicePara* deviceParaList;
	hipFree(ID);
	hipFree(workSpeed);
	hipFree(size);
	hipFree(axis);
	hipFree(direct);
	hipFree(spaceLength);
	hipFree(adjPInCount);
	hipFree(adjPOutCount);
	hipFree(accumAdjPInCount);
	hipFree(accumAdjPOutCount);
	hipFree(adjPointsIn);
	hipFree(adjPointsOut);


	//CargoType* cargoTypeList;				//��������
	hipFree(deviceSum);
	hipFree(linkSum);
	hipFree(accumLinkSum);
	hipFree(deviceLinkList);
	hipFree(totalVolume);


	//�������ӵ���������Ϣ
	hipFree(curBestFitnessVal);
	hipFree(inoutPoints);
	hipFree(strConveyorList);
	hipFree(strConveyorListSum);
	hipFree(curveConveyorList);
	hipFree(curveConveyorListSum);


	//��ǰ��ѵ���������Ϣ
	hipFree(curBestPath_FitnessVal);
	hipFree(curBestPath_InoutPoints);
	hipFree(curBestPath_StrConveyorList);
	hipFree(curBestPath_StrConveyorListSum);
	hipFree(curBestPath_CurveConveyorList);
	hipFree(curBestPath_CurveConveyorListSum);
	hipFree(pointDirectArray);
}

// ��ʼ���������ӣ�û�и���ȫ����ѣ�
//CPU
void PSOOptimizer::InitialAllParticles()
{
	//��CPU�г�ʼ����������
	for (int i = 0; i < particle_num_; ++i)
	{
		InitialParticle(i);
	}

	//�������CPU�������ݿ���
	//for (int i = 0; i < particle_num_; i++)
	//{
	//	for (int j = 0; j < dim_; j++)
	//	{
	//		cout << position_CPU[i * dim_ + j] << ", ";
	//	}
	//	cout << endl;
	//}

	//���Ӵ�CPU->GPU
	hipMemcpy(position_GPU, position_CPU, sizeof(double) * particle_num_ * dim_, hipMemcpyHostToDevice);
	hipMemcpy(velocity_GPU, velocity_CPU, sizeof(double) * particle_num_ * dim_, hipMemcpyHostToDevice);
	hipMemcpy(best_position_GPU, best_position_CPU, sizeof(double) * particle_num_ * dim_, hipMemcpyHostToDevice);
	hipMemcpy(fitness_GPU, fitness_CPU, sizeof(double) * particle_num_ * fitness_count, hipMemcpyHostToDevice);
	hipMemcpy(best_fitness_GPU, best_fitness_CPU, sizeof(double) * particle_num_ * fitness_count, hipMemcpyHostToDevice);
	//����Fitnessֵ GPU
	GetFitness();
	//��ʼ��pbest��gbest CPU
	for (int i = 0; i < particle_num_; ++i)
	{
		// ��ʼ����������λ��
		for (int j = 0; j < dim_; j++)
		{
			best_position_CPU[i * dim_ + j] = position_CPU[i * dim_ + j];
		}
		// ��ʼ�����Ӹ�����ʷ���
		for (int j = 0; j < fitness_count; j++)
		{
			best_fitness_CPU[i * fitness_count + j] = fitness_CPU[i * fitness_count + j];
		}
	}

}

// ��ʼ��Archive����
void PSOOptimizer::InitialArchiveList()
{
	//ͬ�����Ƚ���һ�����
	vector<Particle> particleList(particle_num_, Particle(dim_, fitness_count));
	for (int i = 0; i < particle_num_; i++)
	{
		//particleList[i].position_ = new double[dim_];
		//particleList[i].velocity_ = new double[dim_];
		//particleList[i].best_position_ = new double[dim_];
		for (int j = 0; j < dim_; j++)
		{
			particleList[i].position_[j] = position_CPU[i * dim_ + j];
			particleList[i].velocity_[j] = velocity_CPU[i * dim_ + j];
			particleList[i].best_position_[j] = best_position_CPU[i * dim_ + j];
		}

		//particleList[i].fitness_ = new double[fitness_count];
		//particleList[i].best_fitness_ = new double[fitness_count];
		for (int j = 0; j < fitness_count; j++)
		{
			particleList[i].fitness_[j] = fitness_CPU[i * fitness_count + j];
			particleList[i].best_fitness_[j] = best_fitness_CPU[i * fitness_count + j];
		}
	}
	//vector<Particle> particleList(this->particles_GPU, this->particles_GPU + this->particle_num_);
	Pareto initPareto(particleList);
	this->archive_list = initPareto.GetPareto();
}

// ����Archive���� �������ÿ�����Ӷ����㣬Ҳ����˵��ֻ��Ҫһ���߳̾Ϳ���
// Ϊ�˷��㣬����ʹ��CPU����
void PSOOptimizer::UpdateArchiveList()
{
	//GPU->CPU
	hipMemcpy(position_CPU, position_GPU, sizeof(double) * particle_num_ * dim_, hipMemcpyDeviceToHost);
	hipMemcpy(velocity_CPU, velocity_GPU, sizeof(double) * particle_num_ * dim_, hipMemcpyDeviceToHost);
	hipMemcpy(best_position_CPU, best_position_GPU, sizeof(double) * particle_num_ * dim_, hipMemcpyDeviceToHost);
	hipMemcpy(fitness_CPU, fitness_GPU, sizeof(double) * particle_num_ * fitness_count, hipMemcpyDeviceToHost);
	hipMemcpy(best_fitness_CPU, best_fitness_GPU, sizeof(double) * particle_num_ * fitness_count, hipMemcpyDeviceToHost);

	//���ȣ����㵱ǰ����Ⱥ��pareto�߽磬���߽����Ӽ��뵽�浵archiving��
	//����һ�����
	vector<Particle> particleList(particle_num_, Particle(dim_, fitness_count));
	for (int i = 0; i < particle_num_; i++)
	{
		//particleList[i].position_ = new double[dim_];
		//particleList[i].velocity_ = new double[dim_];
		//particleList[i].best_position_ = new double[dim_];
		for (int j = 0; j < dim_; j++)
		{
			particleList[i].position_[j] = position_CPU[i * dim_ + j];
			particleList[i].velocity_[j] = velocity_CPU[i * dim_ + j];
			particleList[i].best_position_[j] = best_position_CPU[i * dim_ + j];
		}

		//particleList[i].fitness_ = new double[fitness_count];
		//particleList[i].best_fitness_ = new double[fitness_count];
		for (int j = 0; j < fitness_count; j++)
		{
			particleList[i].fitness_[j] = fitness_CPU[i * fitness_count + j];
			particleList[i].best_fitness_[j] = best_fitness_CPU[i * fitness_count + j];
		}
	}
	//vector<Particle> particleList(this->particles_CPU, this->particles_CPU + particle_num_);
	Pareto pareto1(particleList);
	vector<Particle> curParetos = pareto1.GetPareto();
	//��Σ��ڴ浵�и���֧���ϵ���еڶ���ɸѡ�����Ǳ߽�����ȥ��
	vector<Particle> newParetos;
	curParetos.insert(curParetos.end(), this->archive_list.begin(), this->archive_list.end());//�ϲ�cur��ԭArchive
	Pareto pareto2(curParetos);
	vector<Particle> curArchives = pareto2.GetPareto();
	//���µ�ǰarchive
	this->archive_list = curArchives;
}

// ��ʼ��ȫ������
void PSOOptimizer::InitGbest()
{

	GetGbest getG(this->archive_list, this->meshDivCount, this->lower_bound_CPU, this->upper_bound_CPU, this->dim_, this->particle_num_);
	Particle* gbestList = getG.getGbest();
	//����gbestList�Ĳ������ݵ�CPU��Ȼ��GPU
	double* bestFitnessList = (double*)malloc(sizeof(double) * fitness_count * particle_num_);
	double* bestPositionList = (double*)malloc(sizeof(double) * dim_ * particle_num_);
	for (int i = 0; i < particle_num_; i++)
	{
		for (int j = 0; j < fitness_count; j++)
		{
			bestFitnessList[i * fitness_count + j] = gbestList[i].best_fitness_[j];
		}
		for (int k = 0; k < dim_; k++)
		{
			bestPositionList[i * dim_ + k] = gbestList[i].best_position_[k];
		}
	}
	//CPU->GPU
	hipMemcpy(this->all_best_fitness_, bestFitnessList, sizeof(double) * fitness_count * particle_num_, hipMemcpyHostToDevice);
	hipMemcpy(this->all_best_position_, bestPositionList, sizeof(double) * dim_ * particle_num_, hipMemcpyHostToDevice);
}

void PSOOptimizer::GetFitness()
{
	cout << "��ǰ������" << curr_iter_ << endl;
	FitnessFunction << <blockNum, threadsPerBlock >> > (curr_iter_, max_iter_num_, particle_num_, bestParticleIndex,
		/*ProblemParas proParas, �̶������ģ����ù�*/
		DeviceSum, fixedLinkPointSum, fixedUniqueLinkPointSum, vertPointCount, horiPointCount, workShopLength, workShopWidth, convey2DeviceDist, /*double conveyWidth, */
		strConveyorUnitCost, curveConveyorUnitCost, conveyMinDist, /*double conveyMinLength, */conveySpeed, entrancePos, exitPos,
		CargoTypeNum, totalLinkSum,

		/*CargoType* �̶�����*/
		linkSum, accumLinkSum, deviceLinkList, totalVolume,

		/*DevicePara**/
		size, spaceLength, adjPInCount, adjPOutCount, accumAdjPInCount, accumAdjPOutCount,
		totalInPoint, totalOutPoint, adjPointsIn, adjPointsOut,
		/*Particle*/
		dim_, fitness_count, fitness_GPU, position_GPU, /*double* velocity_GPU, double* best_positi on_GPU, double* best_fitness_GPU*/
		/*�洢��������������·��Ϣ*/
		curBestFitnessVal, inoutPSize, inoutPoints, strConveyorList, strConveyorListSum, curveConveyorList, curveConveyorListSum,
		pointDirectArray, globalState);
}

//�ȸĵ�һ��
void PSOOptimizer::UpdateAllParticles()
{
	//���㵱ǰ���Ĺ���ϵ��
	double temp = curr_iter_ / (double)max_iter_num_;
	//temp *= temp;//ϵ���仯
	w_ = wstart_ - (wstart_ - wend_) * temp;
	//���µ�ǰ����������
	//����ֻ�ܴ����洫��ȥ
	UpdateParticle_Kernal << <blockNum, threadsPerBlock >> > (curr_iter_, max_iter_num_, dim_, fitness_count, w_, C1_, C2_, dt_,
		fitness_GPU, position_GPU, velocity_GPU, best_position_GPU, best_fitness_GPU,
		globalState, randomNumList, range_interval_, upper_bound_, lower_bound_, all_best_position_,
		size, spaceLength, workShopLength, workShopWidth);//���allBest��CPU�ģ�
	GetFitness();
	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		//goto Error;
	}
	curr_iter_++;
}

//����Pbest
void PSOOptimizer::UpdatePbest()
{
	//����GPU����
	UpdatePbest_Kernal << <blockNum, threadsPerBlock >> > (dim_, fitness_count, fitness_GPU, position_GPU, velocity_GPU,
		best_position_GPU, best_fitness_GPU, globalState);
}
// ����Gbest GPU
void PSOOptimizer::UpdateGbest()
{
	vector<Particle> tempArchiveL(this->archive_list);
	//GPU->CPU
	//hipMemcpy(particles_CPU, particles_GPU, sizeof(Particle) * particle_num_, hipMemcpyDeviceToHost);
	hipMemcpy(position_CPU, position_GPU, sizeof(double) * particle_num_ * dim_, hipMemcpyDeviceToHost);
	hipMemcpy(velocity_CPU, velocity_GPU, sizeof(double) * particle_num_ * dim_, hipMemcpyDeviceToHost);
	hipMemcpy(best_position_CPU, best_position_GPU, sizeof(double) * particle_num_ * dim_, hipMemcpyDeviceToHost);
	hipMemcpy(fitness_CPU, fitness_GPU, sizeof(double) * particle_num_ * fitness_count, hipMemcpyDeviceToHost);
	hipMemcpy(best_fitness_CPU, best_fitness_GPU, sizeof(double) * particle_num_ * fitness_count, hipMemcpyDeviceToHost);

	hipMemcpy(lower_bound_CPU, lower_bound_, sizeof(double) * dim_, hipMemcpyDeviceToHost);
	hipMemcpy(upper_bound_CPU, upper_bound_, sizeof(double) * dim_, hipMemcpyDeviceToHost);
	GetGbest getG(tempArchiveL, this->meshDivCount, lower_bound_CPU, upper_bound_CPU, this->dim_, this->particle_num_);
	Particle* gbestList = getG.getGbest();

	//GPU
	UpdateGbest_Kernal << <blockNum, threadsPerBlock >> > (fitness_count, dim_, all_best_fitness_, all_best_position_, gbestList);
}

//��ʼ����i������
//�����problemParas����ֱ����CPU��
void PSOOptimizer::InitialParticle(int i)
{
#pragma region ��ʼ��position/veloctiyֵ
	//���������Ȼ����ݳ���������ӵķ�Χ
	for (int j = 2; j < dim_; j += 3)
	{
		//cout << range_interval_CPU[j] << ", ";
		position_CPU[i * dim_ + j] = GetDoubleRand() * range_interval_CPU[j] + lower_bound_CPU[j];
		velocity_CPU[i * dim_ + j] = GetDoubleRand() * range_interval_CPU[j] / 300;
	}
	//cout << endl;
	//���ݳ����޸��豸���½緶Χ&�豸����
	vector<Vector2> deviceSizeCopy;
	for (int i = 0; i < problemParas.DeviceSum; i++)
	{
		deviceSizeCopy.push_back(Vector2(problemParas.size[i].x, problemParas.size[i].y));
	}
	for (int j = 2; j < dim_; j += 3)
	{
		//doubleתint��ת��ΪDirection��Ȼ����ݳ������¼����豸�ߴ�ͳ����
		//Rotate90����Rotate270,�޸�������
		DeviceDirect curDirect = (DeviceDirect)(int)position_CPU[i * dim_ + j];
		if (curDirect == DeviceDirect::Rotate90 || curDirect == DeviceDirect::Rotate270)
		{
			//x��y
			lower_bound_CPU[j - 2] = 0 + problemParas.size[j / 3].y * 0.5 + problemParas.spaceLength[j / 3];
			lower_bound_CPU[j - 1] = 0 + problemParas.size[j / 3].x * 0.5 + problemParas.spaceLength[j / 3];

			upper_bound_CPU[j - 2] = problemParas.workShopLength - problemParas.size[j / 3].y * 0.5 - problemParas.spaceLength[j / 3];
			upper_bound_CPU[j - 1] = problemParas.workShopWidth - problemParas.size[j / 3].x * 0.5 - problemParas.spaceLength[j / 3];

			//size��x��y��Ҫ����
			swap(deviceSizeCopy[j / 3].x, deviceSizeCopy[j / 3].y);
		}
		else
		{
			//x��y
			lower_bound_CPU[j - 2] = 0 + problemParas.size[j / 3].x * 0.5 + problemParas.spaceLength[j / 3];
			lower_bound_CPU[j - 1] = 0 + problemParas.size[j / 3].y * 0.5 + problemParas.spaceLength[j / 3];

			upper_bound_CPU[j - 2] = problemParas.workShopLength - problemParas.size[j / 3].x * 0.5 - problemParas.spaceLength[j / 3];
			upper_bound_CPU[j - 1] = problemParas.workShopWidth - problemParas.size[j / 3].y * 0.5 - problemParas.spaceLength[j / 3];
		}
		range_interval_CPU[j - 2] = upper_bound_CPU[j - 2] - lower_bound_CPU[j - 2];
		range_interval_CPU[j - 1] = upper_bound_CPU[j - 1] - lower_bound_CPU[j - 1];
	}


#pragma region ��ȫ���
	//for (int j = 0; j < dim_; j += 3) {
	//	particles_[i].position_[j] = GetDoubleRand() * range_interval_[j] + lower_bound_[j];
	//	particles_[i].position_[j + 1] = GetDoubleRand() * range_interval_[j + 1] + lower_bound_[j + 1];
	//	particles_[i].velocity_[j] = GetDoubleRand() * range_interval_[j] / 300;
	//	particles_[i].velocity_[j + 1] = GetDoubleRand() * range_interval_[j + 1] / 300;
	//}
#pragma endregion

#pragma region ���Ƿ��ص�Լ��������ֿ���������
//(ÿ��1�ײ���һ������㣬ֻҪ�ҵ�һ�������������ص�Լ�����Ͳ��ã�����Ĭ��Ϊ0
//�µ����������豸�İڷ�˳��
	vector<int> unmakeDeviceIndexVec;
	vector<int> madeDeviceIndexVec;
	for (int i = 0; i < problemParas.DeviceSum; i++)
	{
		unmakeDeviceIndexVec.push_back(i);
	}
	default_random_engine e;

	clock_t startTime, endTime;
	startTime = clock();//��ʱ��ʼ
	while (unmakeDeviceIndexVec.size() > 0)
	{
		endTime = clock();
		if ((static_cast<double>(endTime - startTime) / CLOCKS_PER_SEC) > 1) {
#pragma region ��ȫ���
			for (int j = 0; j < dim_; j += 3) {
				position_CPU[i * dim_ + j] = GetDoubleRand() * range_interval_CPU[j] + lower_bound_CPU[j];
				position_CPU[i * dim_ + j + 1] = GetDoubleRand() * range_interval_CPU[j + 1] + lower_bound_CPU[j + 1];
				velocity_CPU[i * dim_ + j] = GetDoubleRand() * range_interval_CPU[j] / 300;
				velocity_CPU[i * dim_ + j + 1] = GetDoubleRand() * range_interval_CPU[j + 1] / 300;
			}
#pragma endregion
			break;
		}
		//΢�뼶���ȵ����������
		e.seed(GetRamdonSeed());
		uniform_int_distribution<unsigned> u(0, unmakeDeviceIndexVec.size() - 1);
		int randomVecIndex = u(e);
		int randomDeviceIndex = unmakeDeviceIndexVec[randomVecIndex];//�õ��豸��index
		int j = randomDeviceIndex * 3;

		double Xstart = lower_bound_CPU[j];
		double Ystart = lower_bound_CPU[j + 1];

		double tempPositionX = 0;
		double tempPositionY = 0;

		bool findParticle = false;
		while (Ystart <= upper_bound_CPU[j + 1] - 1 && findParticle == false) {//X��YҪ�ڷ�Χ��
			Xstart = lower_bound_CPU[j];
			while (Xstart <= upper_bound_CPU[j] - 1 && findParticle == false) {
				tempPositionX = GetDoubleRand() * 1.0 + Xstart;//�õ�Xstart��Xstart+1֮���һ�������
				tempPositionY = GetDoubleRand() * 1.0 + Ystart;//�õ�Ystart��Ystart+1֮���һ�������
				double halfX = deviceSizeCopy[j / 3].x * 0.5 + problemParas.spaceLength[j / 3];
				double halfY = deviceSizeCopy[j / 3].y * 0.5 + problemParas.spaceLength[j / 3];
				double tempLowX = tempPositionX - halfX;
				double tempUpX = tempPositionX + halfX;
				double tempLowY = tempPositionY - halfY;
				double tempUpY = tempPositionY + halfY;

				bool IsCross = false;
				//��鵱ǰ�豸�Ƿ��������ص�
				for (int k = 0; k < madeDeviceIndexVec.size(); k++)
				{
					int curDeviceIndex = madeDeviceIndexVec[k];
					int curDimIndex = curDeviceIndex * 3;

					double halfX1 = deviceSizeCopy[curDeviceIndex].x * 0.5 + problemParas.spaceLength[curDeviceIndex];
					double halfY1 = deviceSizeCopy[curDeviceIndex].y * 0.5 + problemParas.spaceLength[curDeviceIndex];

					double curLowX = position_CPU[i * dim_ + curDimIndex] - halfX1;
					double curUpX = position_CPU[i * dim_ + curDimIndex] + halfX1;
					double curLowY = position_CPU[i * dim_ + curDimIndex + 1] - halfY1;
					double curUpY = position_CPU[i * dim_ + curDimIndex + 1] + halfY1;
					//�������ص����˳�
					if (IsRangeOverlap(tempLowX, tempUpX, curLowX, curUpX) && IsRangeOverlap(tempLowY, tempUpY, curLowY, curUpY)) {
						IsCross = true;
						break;
					}
				}
				//ȫ�����ص��������Ӹ�ֵ
				if (IsCross == false) {
					findParticle = true;
					position_CPU[i * dim_ + j] = tempPositionX;
					position_CPU[i * dim_ + j + 1] = tempPositionY;
					velocity_CPU[i * dim_ + j] = GetDoubleRand() * range_interval_CPU[j] / 300;
					velocity_CPU[i * dim_ + j + 1] = GetDoubleRand() * range_interval_CPU[j + 1] / 300;

					//����vec
					madeDeviceIndexVec.push_back(randomDeviceIndex);
					unmakeDeviceIndexVec.erase(unmakeDeviceIndexVec.begin() + randomVecIndex);

				}
				Xstart++;
				if (Xstart >= upper_bound_CPU[j] - 1) {
					Ystart++;
				}
			}
		}
	}
#pragma endregion

#pragma endregion


}

