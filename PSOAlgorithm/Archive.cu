#include "hip/hip_runtime.h"
#pragma once
#include "Archive.h"
// ���캯��
MeshCrowd::MeshCrowd(vector<Particle> curArchiveList, int meshDivCount, double* minList, double* maxList, int dim, int ParticleCount)
{
	cout << "����MeshCrowd" << endl;
	this->curArchiveList = curArchiveList;//�浵��������
	this->curArchiveLength = curArchiveList.size();//�浵����

	this->meshDivCount = meshDivCount;//����ȷ����ӣ�Ĭ��Ϊ10
	this->ParticleCount = ParticleCount;//����Ⱥ����

	this->archiveMeshIdList = new int[this->curArchiveLength];//���ӱ��
	this->archiveCrowdList = new int[this->curArchiveLength];//ӵ���Ⱦ������ڼ�¼��ǰ�������������
	for (int i = 0; i < this->curArchiveLength; i++)
	{
		this->archiveMeshIdList[i] = 0;
		this->archiveCrowdList[i] = 0;
	}
	this->gbestParticleList = new Particle[this->ParticleCount];//��ʼ��gbest�����б�

	this->minList = new double[dim];
	this->maxList = new double[dim];
	for (int i = 0; i < dim; i++)
	{
		this->minList[i] = minList[i];
		this->maxList[i] = maxList[i];
	}
	cout << "����MeshCrowd" << endl;
}

// ��������
MeshCrowd::~MeshCrowd()
{
	if (archiveMeshIdList) { delete[]archiveMeshIdList; }
	if (archiveCrowdList) { delete[]archiveCrowdList; }

	if (gbestParticleList) { delete[]gbestParticleList; }
	if (minList) { delete[]minList; }
	if (maxList) { delete[]maxList; }
}

// ��������id
int MeshCrowd::CalMeshId(Particle particle)
{
	//����������id
	//���ȣ���ÿ��ά�Ȱ��յȷ����ӽ��еȷ���ɢ����
	//��ȡ�����ڸ�ά���ϵı�š�����10���ƽ�ÿһ��ά�ȱ�ŵȱ���ӣ�����û��Զ�����mesh_div_num��ֵ�������Զ��壩,�����ֵ
	//Ҳ���ǣ�ÿ��ά���϶���һ��ֵ
	int id = 0;
	for (int i = 0; i < particle.dim_; i++)
	{
		int id_dim = (int)((particle.position_[i] - minList[i]) * meshDivCount / (maxList[i] - minList[i]));
		id = id + id_dim * (pow(meshDivCount, i));
	}
	return id;
}

// ��ÿ�����Ӷ��������ţ������е����ӷŵ���ͬ�������У�
void MeshCrowd::DivideArchiving()
{
	for (int i = 0; i < this->curArchiveLength; i++)
	{
		this->archiveMeshIdList[i] = CalMeshId(curArchiveList[i]);
	}
}

// �����������������ӵ�ӵ����
void MeshCrowd::GetCrowd()
{
	//����һ�����������Ӽ��������ţ����ڸ�������
	vector<int> particleIndexList(this->curArchiveLength);
	//��ʼ��indexlist��ֵ
	for (int i = 0; i < particleIndexList.size(); i++)
	{
		particleIndexList[i] = i;
	}
	while (particleIndexList.size() > 0)
	{
		//��ű�����ѭ������index[0]���Ӿ�����ͬ����id���м���λ
		vector<int> index_same;
		index_same.push_back(particleIndexList[0]);
		for (int i = 1; i < particleIndexList.size(); i++)
		{
			if (this->archiveMeshIdList[particleIndexList[0]] == this->archiveMeshIdList[particleIndexList[i]])
			{
				index_same.push_back(particleIndexList[i]);
			}
		}
		int particleNum = index_same.size();//����������������
		for (int i = 0; i < index_same.size(); i++)//���±�������id�µ��������ӵ�ӵ����
		{
			this->archiveCrowdList[index_same[i]] = particleNum;
			//�ҵ�particleIndexList��index_same[i]��ƥ��λ��
			auto it = find(particleIndexList.begin(), particleIndexList.end(), index_same[i]);
			particleIndexList.erase(it);//ɾ���������������������Ӷ�Ӧ�������ţ������ظ�����
		}
	}

}

// ���캯��
GetGbest::GetGbest(vector<Particle> curArchiveList, int meshDivCount, double* minList, double* maxList, int dim, int ParticleCount) :
	MeshCrowd(curArchiveList, meshDivCount, minList, maxList, dim, ParticleCount)
{
	archiveProbability = new double[this->curArchiveLength];
	for (int i = 0; i < this->curArchiveLength; i++)
	{
		archiveProbability[i] = 0;
	}
	this->DivideArchiving();
	this->GetCrowd();
}

// �������ӱ�ѡ��ĸ���
void GetGbest::GetProbability()
{
	double totalProb = 0.0;
	for (int i = 0; i < this->curArchiveLength; i++)
	{
		archiveProbability[i] = 1.0 / pow(archiveCrowdList[i], 3);
		totalProb += archiveProbability[i];
	}
	for (int i = 0; i < this->curArchiveLength; i++)
	{
		archiveProbability[i] = archiveProbability[i] / totalProb;//��һ�������ӱ�ѡ������ۺ�Ϊ1
	}
}

// ��������
int GetGbest::GetGbestIndex()
{
	double randomProb = rand() % 1000 / (double)1000;//�������С��
	for (int i = 0; i < this->curArchiveLength; i++)
	{
		double subProbTotal = 0.0;
		for (int j = 0; j < i + 1; j++)
		{
			subProbTotal += this->archiveProbability[j];
		}
		if (randomProb <= subProbTotal)
		{
			return i;
		}
	}
}

// ���㵱ǰ����Ⱥ��ÿһ�����ӵ�ȫ�����Ž⼯
Particle* GetGbest::getGbest()
{
	this->GetProbability();//����浵��ÿ�����Ӻ�ӵ������صĸ���
	//����ӵ���ȸߵ����ѡ��gbest
	for (int i = 0; i < this->ParticleCount; i++)
	{
		int gbestIndex = this->GetGbestIndex();
		this->gbestParticleList[i] = this->curArchiveList[gbestIndex];
	}
	return this->gbestParticleList;
}


// �������ӱ�ѡ��ĸ���1
void GetGbest::GetProbability1()
{
	int total = 0;
	for (int i = 0; i < this->curArchiveLength; i++)
	{
		total += this->archiveCrowdList[i];
	}
	for (int i = 0; i < this->curArchiveLength; i++)
	{
		this->archiveProbability[i] = this->archiveCrowdList[i] / total;
	}
}

// ������������ӣ�ӵ���ȸߵ����ӱ�����ĸ���Խ��
vector<int> GetGbest::GetClearIndex(int thresh)
{
	int clearLen = this->curArchiveList.size() - thresh;//��Ҫ���������������
	vector<int> clearIndexList;
	while (clearIndexList.size() < clearLen)
	{
		double randomProb = rand() % 1000 / (double)1000;//�������С��
		for (int i = 0; i < this->curArchiveLength; i++)
		{
			double subTotal = 0.0;
			for (int j = 0; j < i + 1; j++)
			{
				subTotal += this->archiveProbability[j];
			}
			if (randomProb <= subTotal)
			{
				//����i�Ĳ��ӽ�ȥ
				if (find(clearIndexList.begin(), clearIndexList.end(), i) == clearIndexList.end())
				{
					clearIndexList.push_back(i);//��¼����ֵ
					break;
				}
			}
		}
	}
	return clearIndexList;
}

// Clear����
vector<Particle> GetGbest::Clear(int thresh)
{
	this->GetProbability1();
	vector<int> clearIndexList = this->GetClearIndex(thresh);//��Ҫɾ�����±�����
	vector<Particle> resultArchive;
	int curIndex = 0;
	for (auto iter = curArchiveList.begin(); iter != curArchiveList.end(); iter++)
	{
		if (curIndex < clearIndexList.size())
		{
			if (iter != curArchiveList.begin() + clearIndexList[curIndex])
			{
				resultArchive.push_back(*iter);
			}
			else
			{
				curIndex++;
			}
		}
	}
	return resultArchive;
}